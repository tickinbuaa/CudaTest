#include <hip/hip_runtime.h>
#include <memory>

__device__
inline void mac_with_carry(uint64_t &lo, uint64_t &hi, const uint64_t &a, const uint64_t &b, const uint64_t &c) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("GPU calculation input: a = %lx b = %lx c = %lx\n", a, b, c);
    }
    asm("mad.lo.cc.u64 %0, %2, %3, %4;\n\t"
        "madc.hi.u64 %1, %2, %3, 0;\n\t"
        :"=l"(lo), "=l"(hi): "l"(a), "l"(b), "l"(c));
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("GPU calculation result: hi = %lx low = %lx\n", hi, lo);
    }
}

__global__
void test(uint64_t *out, uint32_t num){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= num) {
        return;
    }
    uint64_t a = 0x42737a020c0d6393UL;
    uint64_t b = 0xffffffff00000001UL;
    uint64_t c = 0xc999e990f3f29c6dUL;
    mac_with_carry(out[tid << 1], out[(tid << 1) + 1], a, b, c);
}

int main() {
    uint64_t *d_out;
    uint32_t num = 1;
    hipMalloc(&d_out, num * 2 * sizeof(uint64_t));
    const uint32_t BLOCK_SIZE = 256;
    uint32_t block_num = (num + BLOCK_SIZE - 1) / BLOCK_SIZE;
    test<<<block_num, BLOCK_SIZE>>>(d_out, num);
    hipDeviceSynchronize();
    unsigned __int128 a = 0x42737a020c0d6393UL;
    unsigned __int128 b = 0xffffffff00000001UL;
    unsigned __int128 c = 0xc999e990f3f29c6dUL;
    unsigned __int128 result = a * b + c;
    printf("Cpu result: hi:%lx low:%lx\n", (uint64_t)((result >> 64) & 0xffffffffffffffffUL), (uint64_t)(result & 0xffffffffffffffffUL));
}
